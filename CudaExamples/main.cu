﻿#include "hip/hip_runtime.h"
#include ""
#include "schoolbook/samples_1.h"
#include "schoolbook/samples_2.h"
#include <stdio.h>
#include<conio.h>
#include "../CudaExamples/slides/01_prefixsum.h";
// permette di stabilire se abilitare o meno la modalità console per alcuni degli esempi
int main()
{
    int selection;
    do {
        printf("\n\n");
        printf("SELECT THE EXAMPLE TO RUN:\n");
        printf("1_helloworld\n");
        printf("2_checkDimension\n");
        printf("3_defineGridBlock\n");
        printf("4_sumArraysOnGPUSmallCase\n");
        printf("5_sumArraysOnGPUTimer\n");
        printf("6_checkThreadIndex\n");
        printf("7_matrixSummationOnGPU2DGrid2DBlock\n");
        printf("8_checkDeviceInfo\n");
        printf("9_determiningTheBestGPU\n");
        printf("10_simpleWarpDivergence\n");
        printf("11_reduceInteger\n");
        printf("12_nestedHelloWorld\n");
        printf("13_nestedReduce\n");
        printf("14_nestedReduce2 (to check)\n");
        printf("-----------\n");
        printf("15_globalVariableDeclarationAndModification\n");
        printf("16_simplemMemTransfer\n");
        printf("17_sumArraysZeroCopy\n");
        printf("\n---- SLIDES EXAMPLES ----\n");
        printf("200_prefixsum");
        printf("\n");
        scanf("%d", &selection);
        printf("\n");
        if (selection == 0)
            break;
        switch (selection) {
        case 1:
            helloWorld();
            break;
        case 2:
            checkDimension();
            break;
        case 3:
            defineGridBlock();
            break;
        case 4: 
            sumArraysOnGPUSmallCase();
            break;
        case 5: 
            sumArraysOnGPUTimer();
            break;
        case 6:
            checkThreadIndex();
            break;
        case 7:
            matrixSummationOnGPU2DGrid2DBlock();
            break;
        case 8: 
            checkDeviceInfo();
            break;
        case 9: 
            determiningTheBestGPU();
            break;
        case 10:
            simpleWarpDivergence();
            break;
        case 11: 
            reduceInteger();
            break;
        case 12: 
            nestedHelloWorld();
            break;
        case 13: 
            nestedReduce();
            break;
        case 14: 
            nestedReduce2();
            break;
        case 15:
            globalVariableDeclarationAndModification();
            break;
        case 16: 
            simplemMemTransfer();
            break;
        case 17:
            sumArraysZeroCopy();
            break;
        case 200:
            bool en = false;
            int temp;
            printf("enable console? (0-1)");
            scanf("%d", &temp);
            en = temp;
            prefixSumFirstVersion(en);
            break;
        }
        printf("press a key to continue");
        getch();
    } while (selection > 0);
    return 0;
}
