﻿#include "hip/hip_runtime.h"
#include ""
#include "schoolbook/samples_1.h"
#include "schoolbook/samples_2.h"
#include <stdio.h>
#include<conio.h>
#include "../CudaExamples/slides/01_prefixsum.h";
// permette di stabilire se abilitare o meno la modalità console per alcuni degli esempi
int main()
{
    int selection;
    do {
        printf("\n\n");
        printf("SELECT THE EXAMPLE TO RUN:\n");
        printf("1_helloworld\n");
        printf("2_checkDimension\n");
        printf("3_defineGridBlock\n");
        printf("4_sumArraysOnGPUSmallCase\n");
        printf("5_sumArraysOnGPUTimer\n");
        printf("6_checkThreadIndex\n");
        printf("7_matrixSummationOnGPU2DGrid2DBlock\n");
        printf("8_checkDeviceInfo\n");
        printf("9_determiningTheBestGPU\n");
        printf("10_simpleWarpDivergence\n");
        printf("11_reduceInteger\n");
        printf("12_nestedHelloWorld\n");
        printf("13_nestedReduce\n");
        printf("14_nestedReduce2 (to check)\n");
        printf("-----------\n");
        printf("15_globalVariableDeclarationAndModification\n");
        printf("16_simplemMemTransfer\n");
        printf("17_sumArraysZeroCopy\n");
        printf("\n---- SLIDES EXAMPLES ----\n");
        printf("200_prefixsum\n");
        printf("201_prefixsum (second version)\n");
        printf("202_prefixsum (third version)\n");
        printf("203_prefixsum (fourth version)\n");
        printf("204_prefixsum (fifth version)\n");
        printf("\n");
        scanf("%d", &selection);
        printf("\n");
        if (selection == 0)
            break;
        switch (selection) {
        case 1:
            helloWorld();
            break;
        case 2:
            checkDimension();
            break;
        case 3:
            defineGridBlock();
            break;
        case 4: 
            sumArraysOnGPUSmallCase();
            break;
        case 5: 
            sumArraysOnGPUTimer();
            break;
        case 6:
            checkThreadIndex();
            break;
        case 7:
            matrixSummationOnGPU2DGrid2DBlock();
            break;
        case 8: 
            checkDeviceInfo();
            break;
        case 9: 
            determiningTheBestGPU();
            break;
        case 10:
            simpleWarpDivergence();
            break;
        case 11: 
            reduceInteger();
            break;
        case 12: 
            nestedHelloWorld();
            break;
        case 13: 
            nestedReduce();
            break;
        case 14: 
            nestedReduce2();
            break;
        case 15:
            globalVariableDeclarationAndModification();
            break;
        case 16: 
            simplemMemTransfer();
            break;
        case 17:
            sumArraysZeroCopy();
            break;
        case 200:
            prefixSumFirstVersion();
            break;
        case 201: 
            prefixSumSecondVersion();
            break;
        case 202: 
            prefixSumThirdVersion();
            break;
        case 203:
            prefixSumFourthVersion();
            break;
        case 204: 
            prefixSumFifthVersion();
            break;
        }
        printf("press a key to continue");
        getch();
    } while (selection > 0);
    return 0;
}
